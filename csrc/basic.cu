#include <cassert>
#include <chrono>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <random>

// You may increase this value to test larger matrices
// But it will be slow on CPU
constexpr int MAXN = 1 << 28;
constexpr int TPB = 1 << 8;
void vectorAddCPU(float *a, float *b, float *c, const int N) {
  for (int i = 0; i < N; ++i) {
    c[i] = a[i] + b[i];
  }
}

void initialize(float *a, float *b, const int N) {
  auto gen = std::mt19937(2024);
  auto dis = std::uniform_real_distribution<float>(-1.0, 1.0);
  for (int i = 0; i < N; ++i) {
    a[i] = dis(gen);
  }
  for (int i = 0; i < N; ++i) {
    b[i] = dis(gen);
  }
}

bool compare(float *a, float *b, const int N) {
  for (int i = 0; i < N; ++i) {
    if (std::abs(a[i] - b[i]) > 1e-3) {
      printf("Mismatch at index %d: %f vs %f\n", i, a[i], b[i]);
      return false;
    }
  }
  printf("Results match\n");
  return true;
}

__global__ void vectorAddGPU(float *a, float *b, float *c, const int N) {
  // Implement your vector add kernel here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

int main() {
  float *a, *b, *c;
  a = new float[MAXN];
  b = new float[MAXN];
  c = new float[MAXN];
  initialize(a, b, MAXN);

  // CPU computation
  auto start = std::chrono::high_resolution_clock::now();
  vectorAddCPU(a, b, c, MAXN);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  printf("CPU time: %.3fs\n", elapsed.count());

  // ************** START GPU MEMORY ALLOCATION **************
  size_t size = sizeof(float) * MAXN;
  float *a_d, *b_d, *c_d;
  hipMalloc(&a_d, size);
  hipMalloc(&b_d, size);
  hipMalloc(&c_d, size);
  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);

  // ************** START GPU COMPUTATION **************
  start = std::chrono::high_resolution_clock::now();
  vectorAddGPU<<<MAXN / TPB, TPB>>>(a_d, b_d, c_d, MAXN);
  end = std::chrono::high_resolution_clock::now();

  float *result = new float[MAXN];
  // Copy the result from GPU to CPU
  hipMemcpy(result, c_d, size, hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  if (compare(c, result, MAXN)) {
    std::chrono::duration<double> new_elapsed = end - start;
    printf("GPU time: %.3fs\n", new_elapsed.count());
    printf("Speedup: %.2fx\n", elapsed.count() / new_elapsed.count());
  }
}